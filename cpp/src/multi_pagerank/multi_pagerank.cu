#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <mpi.h>
#include <math.h>
#include <sys/time.h>
#include <getopt.h>
#include <limits.h>
#include <float.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <cudf.h>
//#include <hip/hip_runtime_api.h>

#ifndef __APPLE__
#include <fcntl.h> // for posix_fadvise()
#endif

#ifdef __cplusplus
#define __STDC_FORMAT_MACROS 1
#endif
#include <inttypes.h>

#include "global.h"
#include "phsort.h"
#include "utils.h"
#include "cuda_kernels.h"

#include "utilities/error_utils.h"
#include "graph_utils.cuh"
#include "multi_pagerank.cuh"

#define CHUNK_SIZE	(1024*1024)
#define MAX_LINE	(256)

#define RHS_RANDOM	(0)
#define RHS_CONSTANT	(1)
#define RHS_FILE	(2)

#if LOCINT_SIZE == 8 
#define LOCINT_MAX_CHAR	(20)
#else
#define LOCINT_MAX_CHAR	(11)
#endif

// ~200 Mb buffer
// If smaller the hdd/ssd cache may have an 
// effect even with --no-rcache
#if LOCINT_SIZE == 8
#define IOINT_NUM	(1024*1024*10)
#else
#define IOINT_NUM	(1024*1024*10*2)
#endif
char IOBUF[IOINT_NUM*(LOCINT_MAX_CHAR+1)];

//#define MPR_VERBOSE 1

typedef struct {
	int	code;
	REAL	fp;
	char	*str;
} rhsv_t;

typedef struct {
	LOCINT	*u;
	LOCINT	*v;
	int64_t	ned;
} elist_t;


static spmat_t *createSpmat(int n) {

	spmat_t *m = (spmat_t *)Malloc(sizeof(*m));

	m->firstRow = -1;
	m->lastRow = -1;
	m->intColsNum = 0;
	m->extColsNum = 0;
	m->totToSend = 0;

	m->sendNum = 0;
	m->recvNum = 0;

	m->sendNeigh = NULL;
	m->recvNeigh = NULL;
	m->sendCnts = NULL;
	m->recvCnts = NULL;
	m->sendOffs = NULL;
	m->recvOffs = NULL;

	m->rowsToSend_d = NULL;
#ifdef USE_MAPPED_SENDBUF
	m->sendBuffer_m = NULL;
#else
	m->sendBuffer_d = NULL;
#endif
	m->sendBuffer = NULL;
	m->recvBuffer = NULL;

	m->ncsr = n;
	m->nnz = (LOCINT *)Malloc(n*sizeof(LOCINT));
	m->nrows = (LOCINT *)Malloc(n*sizeof(LOCINT));

	m->roff_d = (LOCINT **)Malloc(n*sizeof(LOCINT *));
	m->rows_d = (LOCINT **)Malloc(n*sizeof(LOCINT *));
	m->cols_d = (LOCINT **)Malloc(n*sizeof(LOCINT *));
	m->vals_d = (REAL **)Malloc(n*sizeof(REAL *));

	memset(m->roff_d, 0, n*sizeof(LOCINT *));
	memset(m->rows_d, 0, n*sizeof(LOCINT *));
	memset(m->cols_d, 0, n*sizeof(LOCINT *));
	memset(m->vals_d, 0, n*sizeof(REAL *));

	m->kthr = (LOCINT (*)[2])Malloc(n*sizeof(LOCINT[2]));
	m->koff = (LOCINT (*)[2])Malloc(n*sizeof(LOCINT[2]));

	return m;
}

static void destroySpmat(spmat_t *m) {

	if (m->sendNeigh) free(m->sendNeigh);
	if (m->recvNeigh) free(m->recvNeigh);
	if (m->sendCnts) free(m->sendCnts);
	if (m->recvCnts) free(m->recvCnts);
	if (m->sendOffs) free(m->sendOffs);
	if (m->recvOffs) free(m->recvOffs);

	if (m->rowsToSend_d) CHECK_CUDA(hipFree(m->rowsToSend_d));
#ifndef USE_MAPPED_SENDBUF
	if (m->sendBuffer_d) CHECK_CUDA(hipFree(m->sendBuffer_d));
#endif
	if (m->sendBuffer) {
		CHECK_CUDA(hipHostUnregister(m->sendBuffer));
		free(m->sendBuffer);
	}
	if (m->recvBuffer) {
		CHECK_CUDA(hipHostUnregister(m->recvBuffer));
		free(m->recvBuffer);
	}

	for(int i = 0; i < m->ncsr; i++) {
		if (m->roff_d[i]) CHECK_CUDA(hipFree(m->roff_d[i]));
		if (m->rows_d[i]) CHECK_CUDA(hipFree(m->rows_d[i]));
		if (m->cols_d[i]) CHECK_CUDA(hipFree(m->cols_d[i]));
		if (m->vals_d[i]) CHECK_CUDA(hipFree(m->vals_d[i]));
	}
	free(m->roff_d);
	free(m->rows_d);
	free(m->cols_d);
	free(m->vals_d);

	free(m->kthr);
	free(m->koff);

	return;
}

static void check_row_overlap(LOCINT first_row, LOCINT last_row, int *exchup, int *exchdown) {

	int		rank, ntask, nr;
	LOCINT 		prevrow, nextrow;
	MPI_Request	request[2];
	
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &ntask);

	nr = 0;
	prevrow = nextrow = -1;

	if (rank > 0)	    MPI_Irecv(&prevrow, 1, LOCINT_MPI, rank-1, TAG(rank-1), MPI_COMM_WORLD, &request[nr++]);
	if (rank < ntask-1) MPI_Irecv(&nextrow, 1, LOCINT_MPI, rank+1, TAG(rank+1), MPI_COMM_WORLD, &request[nr++]);

	if (rank < ntask-1) MPI_Send(&last_row,  1, LOCINT_MPI, rank+1, TAG(rank), MPI_COMM_WORLD);
	if (rank > 0)	    MPI_Send(&first_row, 1, LOCINT_MPI, rank-1, TAG(rank), MPI_COMM_WORLD);

        MPI_Waitall(nr, request, MPI_STATUS_IGNORE);

	*exchup   = (prevrow == first_row);
	*exchdown = (nextrow == last_row);

	return;
}
	
void adjust_row_range(size_t N, LOCINT *first_row, LOCINT *last_row) {

	int	rank, ntask;
	
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &ntask);
    
	MPI_Sendrecv(first_row, 1, LOCINT_MPI, (rank+ntask-1)%ntask, rank,
		     last_row, 1, LOCINT_MPI, (rank+1)%ntask, (rank+1)%ntask,
    		     MPI_COMM_WORLD, MPI_STATUS_IGNORE);
	*last_row = ((rank < ntask-1) ? *last_row : N-1);
	if (rank == 0) *first_row = 0;

	return;	
}

static void postIrecvs(REAL *recvBuffer, int recvNum, int *recvNeigh, int64_t *recvOffs, int64_t *recvCnts,
		       MPI_Request *request, MPI_Comm COMM) {

	int ntask;
	MPI_Comm_size(COMM, &ntask);

	for(int i = 0; i < recvNum; i++) {
		if (recvCnts[i]) {
			MPI_Irecv(recvBuffer + recvOffs[i], recvCnts[i],
				  REAL_MPI, recvNeigh[i], TAG(recvNeigh[i]),
				  COMM, request + i);
		}
	}
	return;
}

static void exchangeDataSingle(REAL *sendBuffer, int sendNeigh, int64_t sendCnts,
			       REAL *recvBuffer, int recvNeigh, int64_t recvCnts,
			       MPI_Request *request, MPI_Comm COMM) {

	int rank, ntask;
	
	MPI_Comm_rank(COMM, &rank);
	MPI_Comm_size(COMM, &ntask);

	if (sendCnts) {
		MPI_Send(sendBuffer, sendCnts,
			 REAL_MPI, sendNeigh, TAG(rank),
			 COMM);
	}
	MPI_Wait(request, MPI_STATUS_IGNORE);
	if (recvCnts) {
		MPI_Irecv(recvBuffer, recvCnts,
			  REAL_MPI, recvNeigh, TAG(recvNeigh),
			  COMM, request);
	}
	return;
}

static inline void cancelReqs(MPI_Request *request, int n) {

        int i;
        for(i = 0; i < n; i++) {
		int flag;
		MPI_Test(request+i, &flag, MPI_STATUS_IGNORE);
		if (!flag) {
			MPI_Cancel(request+i);
			MPI_Wait(request+i, MPI_STATUSES_IGNORE);
		}
	}
        return;
}

static void coo2csr(size_t N, spmat_t *m, elist_t *ein) {

	double tg=0, tdr=0;

	double	min_rt, max_rt;
	double	min_rr, max_rr;
	
	int64_t ned, rbytes=0;

	LOCINT	*u=NULL,   *v=NULL;

	int	EXCH_UP, EXCH_DOWN;
	int	rank, ntask;

	LOCINT	*lastrow_all=NULL;
	LOCINT	*rowsToSend=NULL;

	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &ntask);


	u = ein->u;
	v = ein->v;
	ned = ein->ned;
	
	CHECK_CUDA(hipMemcpy(&m->firstRow, &u[0], sizeof(m->firstRow), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipMemcpy(&m->lastRow, &u[ned-1], sizeof(m->lastRow), hipMemcpyDeviceToHost));


	// quick sanity check on (sorted) edges received as input
	check_row_overlap(m->firstRow, m->lastRow, &EXCH_UP, &EXCH_DOWN);
	if (EXCH_UP || EXCH_DOWN) {
		fprintf(stderr, "Processor %d shares rows with neighboring processors!!!\n", rank);
		exit(EXIT_FAILURE);
	}

	//hipProfilerStart();
	tg = MPI_Wtime();

	// temp data using pool
	LOCINT	*u_d=NULL, *v_d=NULL; // alloc-ed in <remove/keep>_rows_cuda() and
				                // dealloc-ed in get_csr_multi_cuda()

	// remove rows smaller than 1
	//ned = remove_rows_cuda(u, v, ned, &u_d, &v_d);
	
	// simply keep the same input 
	ned = keep_all_rows_cuda(u, v, ned, &u_d, &v_d);

	// quick sanity check
	check_row_overlap(m->firstRow, m->lastRow, &EXCH_UP, &EXCH_DOWN);
	if (EXCH_UP || EXCH_DOWN) {
		fprintf(stderr, "Processor %d shares rows with neighboring processors!!!\n", rank);
		exit(EXIT_FAILURE);
	}
	
	// expand [m->firstRow, m->lastRow] ranges in order to partition [0, N-1]
	// (empty rows outside any [m->firstRow, m->lastRow] range may appear as
	// columns in other rows
	//adjust_row_range(N, &m->firstRow, &m->lastRow);
	
	m->intColsNum = m->lastRow - m->firstRow + 1;


	lastrow_all = (LOCINT *)Malloc(ntask*sizeof(*lastrow_all));
	MPI_Allgather(&m->lastRow, 1, LOCINT_MPI, lastrow_all, 1, LOCINT_MPI, MPI_COMM_WORLD);

	get_csr_multi_cuda(u_d, v_d, ned, lastrow_all, ntask,
			   m->nnz, m->nrows, m->roff_d,
			   m->rows_d, m->cols_d, m->vals_d);
	normalize_cols_multi(m->nnz, m->cols_d, m->vals_d, lastrow_all, m->ncsr, MPI_COMM_WORLD);
	for(int i = 0; i < m->ncsr; i++) {
#if 1
		m->kthr[i][0] = 1024;
		m->kthr[i][1] = 16; // good values: 7, 16
		if (m->kthr[i][1] > m->kthr[i][0]) {
			fprintf(stderr, "[%d] Error with thresholds!!!\n", rank);
			exit(EXIT_FAILURE);
		}
		// sort CSR rows in descending order of length
		sort_csr(m->nnz[i], m->nrows[i], m->kthr[i],
			 m->rows_d+i, m->roff_d+i, m->cols_d+i,
			 m->vals_d+i, m->koff[i]);
#else
		m->koff[i][0] = 0;
		m->koff[i][1] = m->nrows[i];
#endif
	}
	get_extdata_cuda(m->ncsr, m->nnz, m->cols_d, lastrow_all,
			 &m->recvNeigh, &m->recvNum, &m->recvCnts, &m->recvOffs, &m->extColsNum, 
			 &m->sendNeigh, &m->sendNum, &m->sendCnts, &m->sendOffs, &m->totToSend,
			 &rowsToSend, MPI_COMM_WORLD);
	if (m->extColsNum) {
		m->recvBuffer = (REAL *)Malloc(m->extColsNum*sizeof(*m->recvBuffer));
		CHECK_CUDA(hipHostRegister(m->recvBuffer, m->extColsNum*sizeof(*m->recvBuffer), hipHostRegisterMapped));
	}
	if (m->sendNum) {
		m->totToSend = m->sendOffs[m->sendNum-1] + m->sendCnts[m->sendNum-1]; // redundant
		m->sendBuffer = (REAL *)Malloc(m->totToSend*sizeof(*m->sendBuffer));
		CHECK_CUDA(hipHostRegister(m->sendBuffer, m->totToSend*sizeof(*m->sendBuffer), hipHostRegisterMapped));
#ifdef USE_MAPPED_SENDBUF
		CHECK_CUDA(hipHostGetDevicePointer((void **)&(m->sendBuffer_m), m->sendBuffer, 0) );
#else
		CHECK_CUDA(hipMalloc(&m->sendBuffer_d, m->totToSend*sizeof(*m->sendBuffer_d)));
#endif
		CHECK_CUDA(hipMalloc(&m->rowsToSend_d, m->totToSend*sizeof(*m->rowsToSend_d)));
		CHECK_CUDA(hipMemcpy(m->rowsToSend_d, rowsToSend, m->totToSend*sizeof(*m->rowsToSend_d), hipMemcpyHostToDevice));
	}

	relabel_cuda_multi(lastrow_all,
			   m->ncsr,
			   (LOCINT *)m->nrows, m->rows_d,
			   (LOCINT *)m->nnz, m->cols_d,
			   m->totToSend, m->rowsToSend_d, MPI_COMM_WORLD);
	tg = MPI_Wtime()-tg;

	if (!ein) {
		MPI_Reduce(&tdr, &min_rt, 1, MPI_DOUBLE, MPI_MIN, 0, MPI_COMM_WORLD);
		MPI_Reduce(&tdr, &max_rt, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

		double rr = rbytes/(1024.0*1024.0)/tdr;
		MPI_Reduce(&rr, &min_rr, 1, MPI_DOUBLE, MPI_MIN, 0, MPI_COMM_WORLD);
		MPI_Reduce(&rr, &max_rr, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

		MPI_Reduce(rank ? &rbytes : MPI_IN_PLACE, &rbytes, 1, MPI_LONG_LONG, MPI_SUM, 0, MPI_COMM_WORLD);
	}
#if	MPR_VERBOSE
	//hipProfilerStop();
	if (0 == rank) {
		if (!ein) {
			printf("\tread  time: %.4lf secs, %.2lf Mbytes/sec\n", tr, rbytes/(1024.0*1024.0)/tr);
			printf("\t\tmin/max disk read time: %.4lf/%.4lf secs, %.2lf/%.2lf Mbytes/sec\n", min_rt, max_rt, min_rr, max_rr);
		}
		printf("\tgen   time: %.4lf secs\n", tg);
		fflush(stdout);
	}
#endif
	// sanity check (untimed)
	if (m->totToSend) {
		CHECK_CUDA(hipMemcpy(rowsToSend, m->rowsToSend_d, m->totToSend*sizeof(*rowsToSend), hipMemcpyDeviceToHost));
		for(int i = 0; i < m->sendNum; i++) {
			for(int j = 0; j < m->sendCnts[i]; j++) {
				if (rowsToSend[m->sendOffs[i]+j] < 0 || rowsToSend[m->sendOffs[i]+j] > m->intColsNum) {
					fprintf(stderr, "[%d] error: rowsToSend[%" PRId64 "] (%d-th row to send to proc %d) = %" PRILOC " > %" PRId64 "\n",
						rank, m->sendOffs[i]+j, j, m->sendNeigh[i], rowsToSend[m->sendOffs[i]+j], m->intColsNum);
					exit(EXIT_FAILURE);
				}
			}
		}
	}
	if (rowsToSend) free(rowsToSend);
	if (lastrow_all) free(lastrow_all);

	return;
}


static void pagerank_solver(int numIter, REAL c, REAL a, rhsv_t rval, spmat_t *m, REAL *pr) {

	int		i, rank, ntask;
	float		evt;
	double		tg=0, tc=0, t=0;
	double		tspmv[2]={0,0}, tmpi[2]={0,0}, td2h[2]={0,0}, th2d[2]={0,0};
	REAL		*r_h=NULL, *r_d[2]={NULL,NULL}, sum;
	MPI_Request	*reqs=NULL;

	hipStream_t	stream[2];
	hipEvent_t	event[4];

	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &ntask);

	CHECK_CUDA(hipStreamCreate(stream));
	CHECK_CUDA(hipStreamCreate(stream+1));

	CHECK_CUDA(hipEventCreate(event));
	CHECK_CUDA(hipEventCreate(event+1));
	CHECK_CUDA(hipEventCreate(event+2));
	CHECK_CUDA(hipEventCreate(event+3));

	CHECK_CUDA(hipMalloc(r_d+0, (m->intColsNum + MAX(1,m->extColsNum))*sizeof(*r_d[0])));
	CHECK_CUDA(hipMalloc(r_d+1, (m->intColsNum + MAX(1,m->extColsNum))*sizeof(*r_d[1])));

	if (m->recvNum)
		reqs = (MPI_Request *)Malloc(m->recvNum*sizeof(MPI_Request));

	MPI_Barrier(MPI_COMM_WORLD);
	tg = MPI_Wtime();

	// generate RHS
	if (rval.code == RHS_RANDOM) {
		generate_rhs(m->intColsNum, r_d[0]);
	} 
	else { //constant
		setarray(r_d[0], m->intColsNum, rval.fp); 
		CHECK_CUDA(hipDeviceSynchronize());
	}
	MPI_Barrier(MPI_COMM_WORLD);
	tg = MPI_Wtime()-tg;
#if 1
	postIrecvs(m->recvBuffer, m->recvNum, m->recvNeigh, m->recvOffs, m->recvCnts,
		   reqs, MPI_COMM_WORLD);
#endif
	//MPI_Pcontrol(1);
	
	// dangling node experiment
	/*
		REAL *bb=0, *d_leaf_vector=0;
		ALLOC_MANAGED_TRY((void**)&d_leaf_vector, sizeof(REAL) * m->intColsNum, 0);
		ALLOC_MANAGED_TRY ((void**)&bb,    sizeof(REAL) * m->intColsNum, 0);
		REAL randomProbability =  static_cast<REAL>( 1.0/m->intColsNum);
  		cugraph::fill(m->intColsNum, bb, randomProbability);
  		int nn = m->intColsNum;
		cugraph::flag_leaves2(nn, m->nnz[rank], m->cols_d[rank], d_leaf_vector);
		cugraph::update_dangling_nodes(m->intColsNum, d_leaf_vector, c);
	*/
	tc = MPI_Wtime();
	START_RANGE("SPMV_ALL_ITER", 1);
	for(i = 0; i < numIter; i++) {

		int s = i&1;
		int d = s^1;

		START_RANGE("BEFORE_SPMV_LOOP", 0);
#ifdef ASYNC_RED
		reduce_cuda_async(r_d[s], m->intColsNum, &sum, stream[0]);
#else
		sum = reduce_cuda(r_d[s], m->intColsNum);
#endif		
		if (m->totToSend) {
			CHECK_CUDA(hipEventRecord(event[2], stream[1]));
#ifdef USE_MAPPED_SENDBUF
			getSendElems(r_d[s], m->rowsToSend_d, m->totToSend, m->sendBuffer_m, stream[1]);
#else
			getSendElems(r_d[s], m->rowsToSend_d, m->totToSend, m->sendBuffer_d, stream[1]);
			CHECK_CUDA(hipMemcpyAsync(m->sendBuffer,
						   m->sendBuffer_d,
						   m->totToSend*sizeof(*m->sendBuffer),
						   hipMemcpyDeviceToHost, stream[1]));
#endif
			CHECK_CUDA(hipEventRecord(event[3], stream[1]));
		}

#ifdef ASYNC_RED
		CHECK_CUDA(hipStreamSynchronize(stream[0]));
#endif
		t = MPI_Wtime();
		MPI_Allreduce(MPI_IN_PLACE, &sum, 1, REAL_MPI, MPI_SUM, MPI_COMM_WORLD);
		tmpi[0] += MPI_Wtime()-t;

		setarray(r_d[d], m->intColsNum, a*sum, stream[0]);
		END_RANGE;


		START_RANGE("SPMV_LOOP", 1);
		for(int k = 0; k < m->ncsr; k++) {

			int curr = (rank+k) % ntask;

			START_RANGE("SPMV_str0", 2);
			CHECK_CUDA(hipEventRecord(event[0], stream[0]));
			computeSpmvAcc(/* NO ADD TERM */
				       c, m->nrows[curr], m->rows_d[curr],
                            	       m->roff_d[curr], m->cols_d[curr], m->vals_d[curr],
                            	       r_d[s], r_d[d], m->koff[curr], stream[0]);
			CHECK_CUDA(hipEventRecord(event[1], stream[0]));
			// dangling node experiment
			/*
    		float dot_res = cugraph::dot( m->intColsNum, d_leaf_vector, r_d[s]);
    		cugraph::axpy(m->intColsNum, dot_res,  bb,  r_d[d]);
   			cugraph::scal(m->intColsNum, static_cast<REAL>(1.0/cugraph::nrm2(m->intColsNum, r_d[d])) , r_d[d]);
			*/
			START_RANGE("MPI+H2D_str1", 3)
			if (k < m->ncsr-1) {

				START_RANGE("SYNC_str1", 4)
				if (k == 0) {
					// wait for the D2H copy of m->sendBuffer initiated before SPMV loop
					if (m->totToSend) {
						CHECK_CUDA(hipStreamSynchronize(stream[1]));
						CHECK_CUDA(hipEventElapsedTime(&evt, event[2], event[3]));
						td2h[0] += evt / 1000.0; 
					}
				}
				END_RANGE;
			
	 			START_RANGE("MPI", 5)
				//MPI_Barrier(MPI_COMM_WORLD);
				t = MPI_Wtime();
				exchangeDataSingle(m->sendBuffer + m->sendOffs[k], m->sendNeigh[k], m->sendCnts[k],
						   m->recvBuffer + m->recvOffs[k], m->recvNeigh[k], m->recvCnts[k],
						   reqs+k, MPI_COMM_WORLD);
				tmpi[0] += MPI_Wtime()-t;
				END_RANGE;

				START_RANGE("H2D_str1", 6)
				if (m->recvCnts[k]) {
					CHECK_CUDA(hipEventRecord(event[2], stream[1]));
					CHECK_CUDA(hipMemcpyAsync(r_d[s] + m->intColsNum + m->recvOffs[k],
								   m->recvBuffer + m->recvOffs[k],
								   m->recvCnts[k]*sizeof(*r_d[s]),
								   hipMemcpyHostToDevice, stream[1]));
					CHECK_CUDA(hipEventRecord(event[3], stream[1]));

					CHECK_CUDA(hipStreamSynchronize(stream[1]));
					CHECK_CUDA(hipEventElapsedTime(&evt, event[2], event[3]));
					th2d[0] += evt / 1000.0; 
				}
				END_RANGE;
			}
			END_RANGE;

			CHECK_CUDA(hipStreamSynchronize(stream[0]));
			CHECK_CUDA(hipEventElapsedTime(&evt, event[0], event[1]));
			tspmv[0] += evt / 1000.0; 
			END_RANGE;
		}
		END_RANGE;
	}
	//MPI_Pcontrol(0);
	END_RANGE;
	MPI_Barrier(MPI_COMM_WORLD);
	tc = MPI_Wtime()-tc;
	sum = reduce_cuda(r_d[numIter&1], m->intColsNum);
	MPI_Reduce(rank?&sum:MPI_IN_PLACE, &sum, 1, REAL_MPI, MPI_SUM, 0, MPI_COMM_WORLD);

	float loc_nrm_1, glob_nrm1;
	loc_nrm_1= cugraph::nrm1(m->intColsNum,r_d[numIter&1]);
	MPI_Allreduce(&loc_nrm_1, &glob_nrm1, 1, REAL_MPI, MPI_SUM, MPI_COMM_WORLD);

	cugraph::scal(m->intColsNum, (float)1.0/glob_nrm1, r_d[numIter&1]);

#if	MPR_VERBOSE
	{
		char fname[256];
		snprintf(fname, 256, "myresult_%d.txt", rank);
		REAL *r = new REAL[m->intColsNum];
		CHECK_CUDA(hipMemcpy(r, r_d[numIter&1], m->intColsNum*sizeof(*r), hipMemcpyDeviceToHost));
		FILE *fp = fopen(fname, "w");
		for(i = 0; i < m->intColsNum; i++)
			fprintf(fp, "%d %E\n",m->firstRow+i, r[i]);
		fclose(fp);
		delete [] r;
	}
#endif

	MPI_Reduce(td2h, td2h+1, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
	MPI_Reduce(rank?td2h:MPI_IN_PLACE, td2h, 1, MPI_DOUBLE, MPI_MIN, 0, MPI_COMM_WORLD);

	MPI_Reduce(tmpi, tmpi+1, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
	MPI_Reduce(rank?tmpi:MPI_IN_PLACE, tmpi, 1, MPI_DOUBLE, MPI_MIN, 0, MPI_COMM_WORLD);

	MPI_Reduce(th2d, th2d+1, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
	MPI_Reduce(rank?th2d:MPI_IN_PLACE, th2d, 1, MPI_DOUBLE, MPI_MIN, 0, MPI_COMM_WORLD);

	MPI_Reduce(tspmv, tspmv+1, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
	MPI_Reduce(rank?tspmv:MPI_IN_PLACE, tspmv, 1, MPI_DOUBLE, MPI_MIN, 0, MPI_COMM_WORLD);

#if	MPR_VERBOSE
	if (0 == rank) {
		printf("\tgen   time: %.4lf secs\n", tg);
		printf("\tcomp  time: %.4lf secs\n", tc);
		printf("\t\tmin/max  d2h: %.4lf/%.4lf secs\n", td2h[0], td2h[1]);
		printf("\t\tmin/max  mpi: %.4lf/%.4lf secs\n", tmpi[0], tmpi[1]);
		printf("\t\tmin/max  h2d: %.4lf/%.4lf secs\n", th2d[0], th2d[1]);
		printf("\t\tmin/max spmv: %.4lf/%.4lf secs\n", tspmv[0], tspmv[1]);
		printf("PageRank sum: %E\n", sum);
	}
#endif

	cancelReqs(reqs, m->recvNum);

	CHECK_CUDA(hipStreamDestroy(stream[0]));
	CHECK_CUDA(hipStreamDestroy(stream[1]));
	CHECK_CUDA(hipEventDestroy(event[0]));
	CHECK_CUDA(hipEventDestroy(event[1]));

	if (r_h) free(r_h);
	if (reqs) free(reqs);

	hipMemcpy(pr,   r_d[numIter&1],   sizeof(float) * m->intColsNum, hipMemcpyDeviceToDevice);

	if (r_d[0]) CHECK_CUDA(hipFree(r_d[0]));
	if (r_d[1]) CHECK_CUDA(hipFree(r_d[1]));

	cudaCheckError();
	return;
}

// Perform gdf input check
// Make local elist_t point to local GDF data 
// No copy
gdf_error load_gdf_input (const gdf_column *src_indices, 
       					  const gdf_column *dest_indices,
       					  elist_t *el) {

  GDF_REQUIRE( src_indices->size == dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((src_indices->dtype == GDF_INT32) || (src_indices->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY ); 

  el->u = (LOCINT*)src_indices->data; 
  el->v = (LOCINT*)dest_indices->data; 
  el->ned = src_indices->size; 
  return GDF_SUCCESS;
}

// Setup local gdf output
// column gdf_v_idx contains global vertex IDs
// column gdf_pr contains corresponding pr
// No copy
gdf_error fill_gdf_output (spmat_t *m, 
						   REAL *pr,
       					   gdf_column *gdf_v_idx, 
       					   gdf_column *gdf_pr) {

if (gdf_v_idx->dtype == GDF_INT64)
 cugraph::sequence<int64_t>(m->intColsNum,(int64_t*)gdf_v_idx->data,(int64_t)m->firstRow);
else
 cugraph::sequence<int>(m->intColsNum,(int*)gdf_v_idx->data,(int)m->firstRow);
    int	rank;

	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

std::cout<< rank<<" "<<m->intColsNum<<std::endl;
  CHECK_CUDA(hipMemcpy(gdf_pr->data, pr, m->intColsNum*sizeof(float), hipMemcpyDeviceToDevice));

  return GDF_SUCCESS;
}

// coo to csr
// spmat_t is a custom structure for distributed csr matriices in PRBench
void gdf_multi_coo2csr_t(size_t N, const gdf_column *src_indices, const gdf_column *dest_indices, spmat_t *m) {
	elist_t * el = (elist_t *)Malloc(sizeof(*el));
	load_gdf_input(dest_indices, src_indices, el);
	coo2csr(N, m, el);
	if (el) free(el); //just free the structure
}



//Build a CSR matrix and solve Pagerank
gdf_error gdf_multi_pagerank_impl (const size_t global_v, const gdf_column *src_indices, const gdf_column *dest_indices, 
	                         gdf_column *v_idx, gdf_column *pagerank, const float damping_factor, const int max_iter) {
	GDF_REQUIRE( ((v_idx->dtype == GDF_INT32) || (v_idx->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
	GDF_REQUIRE((pagerank->dtype == GDF_FLOAT32), GDF_UNSUPPORTED_DTYPE );

    int	rank, ntask;
	rhsv_t	rval = {RHS_CONSTANT, REAL(1.0)/REAL(global_v), NULL};
	REAL a = (REALV(1.0)-REAL(damping_factor))/((REAL)global_v);
	
	//setup 
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &ntask);
	MPI_Barrier(MPI_COMM_WORLD);
	init_cuda();
	spmat_t *m = createSpmat(ntask);
    REAL* pr = nullptr;

    //coo2csr transposed
	gdf_multi_coo2csr_t(global_v, src_indices, dest_indices, m);
	cudaCheckError();
	//allocate local result
	hipMalloc(&pr,m->intColsNum*sizeof(float));
	//solve
	pagerank_solver(max_iter, damping_factor, a, rval, m, pr);

	//store the local result in gdf_columns
	fill_gdf_output(m, pr, v_idx, pagerank);

	//cleanup
	if (rval.str) free(rval.str);
	hipFree(pr);
	destroySpmat(m);
	cleanup_cuda();

	return GDF_SUCCESS;
}