#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Pagerank solver tests
// Author: Alex Fender afender@nvidia.com

#include <mpi.h>
#include <algorithm>
#include "gtest/gtest.h"
#include <cugraph.h>
#include "hip/hip_runtime_api.h"
#include "test_utils.h"

TEST(MultiPagerank, imb32_32B_2ranks)
{
  int rank, ntask;
  MPI_Comm_size(MPI_COMM_WORLD, &ntask);
  ASSERT_EQ(ntask,3) << "This test works for 3 MPI processes"<< "\n";
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  size_t loc_v, loc_e, global_v = 32;
  float damping_factor=0.85;
  int max_iter=30;


  std::vector<int> src_h, dest_h, v_idx;
  std::vector<float> pagerank, nx_ref;


  // This input data was generated from PRbench code
  // ibm data set is split between rank 0,1,2 so they have a similar number of edges
  // Same destinations (keys) cannot be on 2 partitions 
  if(rank == 0) {
    loc_v = 9;
    loc_e = 45;
    src_h={0,1,2,3,6,25,0,1,8,20,27,1,2,5,7,8,28,2,3,4,11,2,4,22,26,0,5,15,2,6,13,20,30,0,7,11,16,26,6,8,9,12,18,22,26};
    dest_h={0,0,0,0,0,0,1,1,1,1,1,2,2,2,2,2,2,3,3,3,3,4,4,4,4,5,5,5,6,6,6,6,6,7,7,7,7,7,8,8,8,8,8,8,8};
    nx_ref={0.0438673505,0.0448996131,0.0529609128,0.0215618329,0.0214298682,0.0181317262,0.0381081597,0.0272124501,0.0756169741};
  }
  if (rank== 1) {
    loc_v = 11;
    loc_e = 39;
    src_h={0,9,10,20,22,24,26,1,10,14,17,28,5,11,23,10,12,2,13,1,14,19,3,15,21,3,15,16,5,9,17,19,29,0,18,25,7,15,19};
    dest_h={9,9,9,9,9,9,9,10,10,10,10,10,11,11,11,12,12,13,13,14,14,14,15,15,15,16,16,16,17,17,17,17,17,18,18,18,19,19,19};
    nx_ref={0.0746216296,0.0505499903,0.0124205849,0.0195928545,0.0143924609,0.0245432364,0.0195166656,0.0170370551,0.0582662552,0.0286961279,0.0230860265};
  }
  if (rank== 2) {
    loc_v = 12;
    loc_e = 42;
    src_h={2,20,31,10,21,1,16,20,22,11,23,25,5,14,17,23,24,12,17,21,25,4,23,25,26,8,27,2,4,26,28,31,11,16,22,29,12,13,30,23,27,31};
    dest_h={20,20,20,21,21,22,22,22,22,23,23,23,24,24,24,24,24,25,25,25,25,26,26,26,26,27,27,28,28,28,28,28,29,29,29,29,30,30,30,31,31,31};
    nx_ref={0.0197781389,0.0215293575,0.0217240197,0.0149364236,0.0515557942,0.0329307776,0.0202471120,0.0289620096,0.0332623858,0.0245434036,0.0224849487,0.0215338530};
  }

  pagerank.resize(loc_v);
  v_idx.resize(loc_v);

  //Check input sizes
  ASSERT_EQ(src_h.size(),dest_h.size());
  ASSERT_EQ(src_h.size(),loc_e);
  ASSERT_EQ(nx_ref.size(),loc_v);
  ASSERT_EQ(pagerank.size(),loc_v);
  ASSERT_EQ(v_idx.size(),loc_v);

  gdf_column *col_src = new gdf_column, 
             *col_dest = new gdf_column, 
             *col_pagerank = new gdf_column, 
             *col_vidx = new gdf_column;

  create_gdf_column(pagerank, col_pagerank);
  create_gdf_column(v_idx, col_vidx);
  //create_gdf_column(src_h, col_src);
  //create_gdf_column(dest_h, col_dest);

  //Check input col sizes
  ASSERT_EQ(col_src->size,loc_e);
  ASSERT_EQ(col_dest->size,loc_e);
  //ASSERT_EQ(col_pagerank->size,loc_v);
  //ASSERT_EQ(col_vidx->size,loc_v);

  ASSERT_EQ(gdf_multi_pagerank (global_v, col_src, col_dest, col_vidx, col_pagerank, damping_factor, max_iter),GDF_SUCCESS);

  std::vector<float> calculated_res(loc_v);
  CUDA_RT_CALL(hipMemcpy(&calculated_res[0],   col_pagerank->data,   sizeof(float) * loc_v, hipMemcpyDeviceToHost));

  std::vector<int> calculated_idx(loc_v);
  CUDA_RT_CALL(hipMemcpy(&calculated_idx[0],   col_vidx->data,   sizeof(int) * loc_v, hipMemcpyDeviceToHost));
  
  float err;
  int n_err = 0;
  for (int i = 0; i < loc_v; i++)
  {
      std::cout<< rank<<" " << calculated_idx[i]<<" " <<nx_ref[i]<<" "<<calculated_res[i]<<std::endl;

      err = fabs(nx_ref[i] - calculated_res[i]);
      if (err> 1e-6f)
      {
          n_err++;
      }
  }
  MPI_Barrier(MPI_COMM_WORLD);

  if (n_err)
  {
      EXPECT_LE(n_err, 0); 
  }

  gdf_col_delete(col_src);
  gdf_col_delete(col_dest);
  gdf_col_delete(col_pagerank);
  gdf_col_delete(col_vidx);
}

int main(int argc, char **argv)  {

  srand(42);
  ::testing::InitGoogleTest(&argc, argv);
  MPI_Init(&argc, &argv);
  
  int r = RUN_ALL_TESTS();
  MPI_Finalize();
  return r;
}

